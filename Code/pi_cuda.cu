#include "hip/hip_runtime.h"
#include <stdio.h>

#define ITERATIONS 96000000
const int threads = 1024; // nVidia's optimal block size should be muptiple of 32

// Synchronous error checking call. Enable with nvcc -DDEBUG
inline void checkCUDAError(const char *fileName, const int line)
{ 
    #ifdef DEBUG
        cudaThreadSynchronize();
        cudaError_t error = cudaGetLastError();
        if(error != cudaSuccess){
        printf("Error at %s: line %i: %s\n", fileName, line, cudaGetErrorString(error));
        exit(-1);
        }
    #endif
}
__global__ void integrateSimple(float *sum)
{
    __shared__ float ssums[threads];
    // Each thread computes its own sum. 
    int global_idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(global_idx < ITERATIONS){
        float step = 1.0f / ITERATIONS;
        float x = (global_idx + 0.5f) * step;
        ssums[threadIdx.x] = 4.0f / (1.0f + x * x);
    }
    else{
    ssums[threadIdx.x] = 0.0f;
    }
    // The 1st thread will gather all sums from all other threads of this block into one
	 __syncthreads();
    if(threadIdx.x == 0)
    {
    float local_sum = 0.0f;
        for(int i = 0; i < threads; ++i)
        {
        local_sum += ssums[i];
        }
        sum[blockIdx.x] = local_sum;
        }
}
int main()
{
    const float PI = 3.14159265358979323846264;
    int deviceCount = 0;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    if (error != hipSuccess)
    {
        printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error, hipGetErrorString(error));
        return 1;
    }
    deviceCount == 0 ? printf("There are no available CUDA device(s)\n") : 
                                                printf("%d CUDA Capable device(s) detected\n" , deviceCount);
    /*--------- Simple Kernel ---------*/
    int blocks = (ITERATIONS + threads - 1) / threads;
    float *sum_d;
    float step = 1.0f / ITERATIONS;

	for (int i = 0; i < 5000; ++i)
    {   // Allocate device memory
        hipMallocManaged((void **)&sum_d, blocks * sizeof(float));
        // CUDA events needed to measure execution time
        hipEvent_t start, stop;
        float gpuTime;
        // Start timer
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
        // calculate pi 
        integrateSimple<<<blocks, threads>>>(sum_d);
        hipDeviceSynchronize(); // wait until the kernel execution is completed
        checkCUDAError(__FILE__, __LINE__);
        // Stop timer
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&gpuTime, start, stop);
        // Sum result on host
        float MyPI = 0.0f;
        for (int i = 0; i < blocks; i++)
        {
            MyPI += sum_d[i];
        }
        MyPI *= step;
        hipFree(sum_d);
        /*printf("\n======================================\n\n");*/
        printf("%.23lf,%.23lf,%f", MyPI, fabs(MyPI - PI), gpuTime/1000);
        printf("\n");
    }
    // Reset Device
    hipDeviceReset();
    return 0;
}


